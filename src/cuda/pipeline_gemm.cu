#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>

#ifndef __HIPCC__
#define __HIPCC__
#define __HAHA__
#endif

#include <cooperative_groups/memcpy_async.h>

#ifdef __HAHA__
#undef __HIPCC__
#endif
#include <cuda/pipeline>

#ifndef __HIPCC__
#include "hip/hip_runtime.h"
#include ""
void __syncthreads(); // workaround __syncthreads warning
#endif

#include <iostream>
const size_t BLOCK_SIZE = 16; // we assume that every block has equal blockDim.x and blockDim.y
const size_t BLOCK_M = 128;   // These const values decide how many thing a thread compute and the amount of shared memory to allocate.
const size_t BLOCK_N = 128;
const size_t BLOCK_K = 8; // don't set 64 here, it will cause bank conflict and lower occupancy.
const size_t BLOCK_M_COMPUTE = BLOCK_M / BLOCK_SIZE;
const size_t BLOCK_N_COMPUTE = BLOCK_N / BLOCK_SIZE;
const size_t BLOCK_K_COMPUTE = BLOCK_K / BLOCK_SIZE;

const int shared_memory_A = BLOCK_M * BLOCK_K;
const int shared_memory_B = BLOCK_N * BLOCK_K;
const int shared_memory_element = shared_memory_A + shared_memory_B;
const int shared_memory_size = shared_memory_element * sizeof(float); // shared memory to use.
#define colM(a, i, j, lda) a[((j) * (lda)) + (i)]
#define rowM(a, i, j, lda) a[(j) + (i) * (lda)]

__forceinline__ __device__ auto convertColIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subM, n = idx % subM;
    return begin + m + n * N;
}

__forceinline__ __device__ auto convertRowIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subN, n = idx % subN;
    return begin + m * N + n;
}

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    const size_t baseX = blockIdx.x * blockDim.x * BLOCK_M_COMPUTE;
    const size_t baseY = blockIdx.y * blockDim.y * BLOCK_N_COMPUTE;

    const int moveNum = shared_memory_element / (BLOCK_SIZE * BLOCK_SIZE) / 2;
    const size_t baseIdx = threadIdx.x * blockDim.x + threadIdx.y;

    auto block = cooperative_groups::this_thread_block();

    float c[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};

    constexpr size_t stage_count = 2;
    __shared__ cuda::pipeline_shared_state<cuda::thread_scope_block, stage_count> shared_state;
    auto pipeline = cuda::make_pipeline(block, &shared_state);

    __shared__ float subA[stage_count][BLOCK_M * BLOCK_K];
    __shared__ float subB[stage_count][BLOCK_N * BLOCK_K];

    pipeline.producer_acquire();
#pragma unroll
    for (int idx = 0; idx < moveNum; idx++)
    {
        cuda::memcpy_async(block, subA[0] + baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, convertColIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, A + baseX * K, BLOCK_M, BLOCK_K, K), sizeof(float), pipeline);
        cuda::memcpy_async(block, subB[0] + baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, convertRowIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, B + baseY, BLOCK_K, BLOCK_N, N), sizeof(float), pipeline);
        // subA[0][baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = *convertColIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, A + baseX * K, BLOCK_M, BLOCK_K, K);
        // subB[0][baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = *convertRowIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, B + baseY, BLOCK_K, BLOCK_N, N);
    }
    pipeline.producer_commit();

    for (int i = BLOCK_K; i < K; i += BLOCK_K)
    {
        size_t copy_stage_idx = (i / BLOCK_K) % 2;
        size_t compute_stage_idx = (i / BLOCK_K - 1) % 2;

        pipeline.producer_acquire();
#pragma unroll
        for (int idx = 0; idx < moveNum; idx++)
        {
            cuda::memcpy_async(block, subA[0] + baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, convertColIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, A + baseX * K, BLOCK_M, BLOCK_K, K), sizeof(float), pipeline);
            cuda::memcpy_async(block, subB[0] + baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, convertRowIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, B + baseY, BLOCK_K, BLOCK_N, N), sizeof(float), pipeline);
            // subA[copy_stage_idx][baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = *convertColIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, A + baseX * K + i, BLOCK_M, BLOCK_K, K);
            // subB[copy_stage_idx][baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE] = *convertRowIdx(baseIdx + idx * BLOCK_SIZE * BLOCK_SIZE, B + baseY + i * N, BLOCK_K, BLOCK_N, N);
        }
        pipeline.producer_commit();

        pipeline.consumer_wait();
#pragma unroll(4)
        for (int ii = 0; ii < BLOCK_K; ii++)
        {
            float regB[BLOCK_M_COMPUTE]; // hopefully, these should reside in register.
#pragma unroll
            for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
            {
                regB[cpj] = subB[compute_stage_idx][threadIdx.y * BLOCK_N_COMPUTE + cpj + BLOCK_N * ii];
            }
#pragma unroll
            for (int cpi = 0; cpi < BLOCK_M_COMPUTE; cpi++)
            {
                float regA = subA[compute_stage_idx][(threadIdx.x * BLOCK_M_COMPUTE + cpi) + ii * BLOCK_M];
#pragma unroll
                for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
                {
                    c[cpi * BLOCK_M_COMPUTE + cpj] += regA * regB[cpj];
                }
            }
        }
        pipeline.consumer_release();
    }

    pipeline.consumer_wait();
#pragma unroll(4)
    for (int ii = 0; ii < BLOCK_K; ii++)
    {
        float regB[BLOCK_M_COMPUTE]; // hopefully, these should reside in register.
#pragma unroll
        for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
        {
            regB[cpj] = subB[(K / BLOCK_K - 1) % 2][threadIdx.y * BLOCK_N_COMPUTE + cpj + BLOCK_N * ii];
        }
#pragma unroll
        for (int cpi = 0; cpi < BLOCK_M_COMPUTE; cpi++)
        {
            float regA = subA[(K / BLOCK_K - 1) % 2][(threadIdx.x * BLOCK_M_COMPUTE + cpi) + ii * BLOCK_M];
#pragma unroll
            for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
            {
                c[cpi * BLOCK_M_COMPUTE + cpj] += regA * regB[cpj];
            }
        }
    }
    pipeline.consumer_release();

    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
        for (int j = 0; j < BLOCK_N_COMPUTE; j++)
            C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] = 0 * C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] + alpha * c[i * BLOCK_M_COMPUTE + j];
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}
