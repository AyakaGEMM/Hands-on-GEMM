#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <hipcub/hipcub.hpp>
#include <cuda_help_func.hpp>

#ifndef __HIPCC__
#include "hip/hip_runtime.h"
#include ""
void __syncthreads(); // workaround __syncthreads warning
void __syncwarp();
#endif

#include <iostream>

enum quantType
{
    MIN_MAX,
    PER_COL,
    PER_ROW
};

class quantMatrixHolder
{
    int8_t *_data;
    size_t _M, _N, _size;
    float *_scales;
    int32_t *_zeroPoints;
    quantType _qt;
    int32_t *_sums;

public:
    quantMatrixHolder() : _data(nullptr), _scales(nullptr), _zeroPoints(nullptr), _sums(nullptr), _M(0), _N(0), _size(0), _qt(MIN_MAX) {}
    ~quantMatrixHolder()
    {
        hipFree(_data);
        hipFree(_scales);
        hipFree(_zeroPoints);
        hipFree(_sums);
    }

    quantMatrixHolder(size_t M, size_t N, quantType qt) : _qt(qt)
    {
        hipMalloc(&_data, M * N);
        if (_qt == MIN_MAX)
        {
            hipMalloc(&_scales, sizeof(float));
            hipMalloc(&_zeroPoints, sizeof(int32_t));
            hipMalloc(&_sums, sizeof(int32_t) * N);
        }
        else if (_qt == PER_COL)
        {
            hipMalloc(&_scales, sizeof(float) * N);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * N);
            hipMalloc(&_sums, sizeof(int32_t) * N);
        }
        else if (_qt == PER_ROW)
        {
            hipMalloc(&_scales, sizeof(float) * M);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * M);
            hipMalloc(&_sums, sizeof(int32_t) * M);
        }
        _M = M;
        _N = N;
        _size = M * N;
    }

    void resize(size_t M, size_t N)
    {
        if (M * N >= _size)
        {
            _size = M * N * 2;
            hipFree(_data);
            hipMalloc(&_data, _size * sizeof(int8_t));
        }

        if (_qt == PER_COL && N >= _N)
        {
            hipFree(_scales);
            hipFree(_zeroPoints);
            hipFree(_sums);
            hipMalloc(&_scales, sizeof(float) * N);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * N);
            hipMalloc(&_sums, sizeof(int32_t) * N);
        }
        else if (_qt == PER_ROW && M >= _M)
        {
            hipFree(_scales);
            hipFree(_zeroPoints);
            hipFree(_sums);
            hipMalloc(&_scales, sizeof(float) * M);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * M);
            hipMalloc(&_sums, sizeof(int32_t) * M);
        }
        else if (_qt == MIN_MAX)
        {
            if (_scales == nullptr)
            {
                hipMalloc(&_scales, sizeof(float));
                hipMalloc(&_zeroPoints, sizeof(int32_t));
            }
            if (N >= _N)
            {
                hipFree(_sums);
                hipMalloc(&_sums, sizeof(int32_t) * N);
            }
        }
        _M = M;
        _N = N;
    }

    void showStatus()
    {
        void *hostMem = malloc(_size * sizeof(float));

        checkCudaErrors(hipMemcpy(hostMem, _data, sizeof(int8_t) * _size, hipMemcpyDeviceToHost));
        std::cout << "Quant Result: " << std::endl;
        for (int i = 0; i < _M; i++)
        {
            for (int j = 0; j < _N; j++)
            {
                std::cout << int16_t(reinterpret_cast<int8_t *>(hostMem)[i * _N + j]) << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        if (_qt == MIN_MAX)
        {
            checkCudaErrors(hipMemcpy(hostMem, _zeroPoints, sizeof(int32_t), hipMemcpyDeviceToHost));
            std::cout << "Zero Point: " << reinterpret_cast<int32_t *>(hostMem)[0] << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _scales, sizeof(float), hipMemcpyDeviceToHost));
            std::cout << "Scale: " << reinterpret_cast<float *>(hostMem)[0] << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _sums, sizeof(int32_t) * _M, hipMemcpyDeviceToHost));
            for (int i = 0; i < _M; i++)
            {
                std::cout << reinterpret_cast<int32_t *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
        }
        else if (_qt == PER_COL)
        {
            checkCudaErrors(hipMemcpy(hostMem, _zeroPoints, sizeof(int32_t) * _N, hipMemcpyDeviceToHost));
            std::cout << "Zero Point: " << std::endl;
            for (int i = 0; i < _N; i++)
            {
                std::cout << reinterpret_cast<int32_t *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _scales, sizeof(float) * _N, hipMemcpyDeviceToHost));
            std::cout << "Scale: " << std::endl;
            for (int i = 0; i < _N; i++)
            {
                std::cout << reinterpret_cast<float *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _sums, sizeof(int32_t) * _N, hipMemcpyDeviceToHost));
            for (int i = 0; i < _N; i++)
            {
                std::cout << reinterpret_cast<int32_t *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
        }
        else if (_qt == PER_ROW)
        {
            checkCudaErrors(hipMemcpy(hostMem, _zeroPoints, sizeof(int32_t) * _M, hipMemcpyDeviceToHost));
            std::cout << "Zero Point: " << std::endl;
            for (int i = 0; i < _M; i++)
            {
                std::cout << reinterpret_cast<int32_t *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _scales, sizeof(float) * _M, hipMemcpyDeviceToHost));
            std::cout << "Scale: " << std::endl;
            for (int i = 0; i < _M; i++)
            {
                std::cout << reinterpret_cast<float *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
            checkCudaErrors(hipMemcpy(hostMem, _sums, sizeof(int32_t) * _M, hipMemcpyDeviceToHost));
            for (int i = 0; i < _M; i++)
            {
                std::cout << reinterpret_cast<int32_t *>(hostMem)[i] << " ";
            }
            std::cout << std::endl;
        }
    }

    size_t size()
    {
        return _size;
    }

    int8_t *dataPtr()
    {
        return _data;
    }

    auto sumsPtr()
    {
        return _sums;
    }

    auto scalesPtr()
    {
        return _scales;
    }

    auto zeroPointsPtr()
    {
        return _zeroPoints;
    }

    void setQuantType(quantType qt)
    {
        hipFree(_scales);
        hipFree(_zeroPoints);
        hipFree(_sums);
        _qt = qt;
        if (_qt == MIN_MAX)
        {
            hipMalloc(&_scales, sizeof(float));
            hipMalloc(&_zeroPoints, sizeof(int32_t));
            hipMalloc(&_sums, sizeof(int32_t) * _M);
        }
        else if (_qt == PER_COL)
        {
            hipMalloc(&_scales, sizeof(float) * _N);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * _N);
            hipMalloc(&_sums, sizeof(int32_t) * _N);
        }
        else if (_qt == PER_ROW)
        {
            hipMalloc(&_scales, sizeof(float) * _M);
            hipMalloc(&_zeroPoints, sizeof(int32_t) * _M);
            hipMalloc(&_sums, sizeof(int32_t) * _M);
        }
    }

    quantMatrixHolder(const quantMatrixHolder &) = delete;
    quantMatrixHolder(const quantMatrixHolder &&) = delete;
    quantMatrixHolder &operator=(const quantMatrixHolder &) = delete;
    quantMatrixHolder &operator=(const quantMatrixHolder &&) = delete;
};

template <typename T, typename S>
__device__ S climp(const T &a)
{
    T min = std::numeric_limits<S>::min(), max = std::numeric_limits<S>::max();
    return std::min(std::max(min, a), max);
}

// develop in progress functions
template <typename T> // naive min max kernel need optimize further
__device__ __forceinline__ void findMinMax(const T *from, T &min, T &max, int32_t N)
{
    const T *to = from + N;
#pragma unroll
    while (from < to)
    {
        T next = *(from++);
        min = (min <= next) ? min : next;
        max = (max <= next) ? next : max;
    }
}

template <typename T> // naive min max kernel need optimize further
__device__ __forceinline__ void getSums(const T *from, T &sum, T &psum, int32_t M, int32_t N)
{
    const T *to = from + M * N;
#pragma unroll
    while (from < to)
    {
        T next = *from;
        from += N;
        sum += next;
        psum += next * next;
    }
}

template <const int BLOCK_THREADS, const int workPerThread>
__global__ void quantInput(
    const float *__restrict__ input,
    const int M,
    const int N,
    const float *max,
    const float *min,
    std::int8_t *__restrict__ output,
    float &scale,
    std::int32_t &zeroPoint,
    std::int32_t *__restrict__ sums)
{
    using namespace hipcub;

    constexpr float kEpsilon = 1e-8f;
    const int64_t baseIdx = (int)blockIdx.x * blockDim.x + threadIdx.x;

    if (baseIdx * workPerThread < N)
    {
        const float *baseA = input + baseIdx * workPerThread * N;
        auto baseOutput = output + baseIdx * workPerThread * N;

        const float range = *max - *min;

        auto Tscale = range / 255;
        const auto invScale = 255.0f / (range + kEpsilon);
        auto TzeroPoint = int32_t(std::nearbyintf(127 - *max * invScale));

        float sum[workPerThread] = {};

#pragma unroll
        for (std::size_t col = 0; col < workPerThread * N && col + baseIdx * workPerThread * N < M * N; ++col)
        {
            baseOutput[col] = climp<float, int8_t>(std::nearbyintf(baseA[col] * invScale + TzeroPoint));
            sum[col / N] += baseOutput[col];
        }
#pragma unroll
        for (int i = 0; i < workPerThread; i++)
        {
            sums[baseIdx * workPerThread + i] = sum[i];
        }

        if (baseIdx == 0)
        {
            scale = Tscale;
            zeroPoint = TzeroPoint;
        }
    }
}

template <int BLOCK_THREADS, int workPerThread>
__global__ void quantWeight(
    const float *__restrict__ input,
    const int M,
    const int N,
    std::int8_t *__restrict__ output,
    float *__restrict__ scales,
    std::int32_t *__restrict__ zeroPoints,
    std::int32_t *__restrict__ sums)
{
    using namespace hipcub;

    constexpr float kEpsilon = 1e-8f;

    const int64_t baseIdx = blockIdx.x * blockDim.x + threadIdx.x;
    auto baseOutput = output + baseIdx * workPerThread;
    float sumCols[workPerThread], psumCols[workPerThread];
#pragma unroll
    for (int i = 0; i < workPerThread; i++)
    {
        sumCols[i] = psumCols[i] = 0;
    }

#pragma unroll
    for (int i = 0; i + baseIdx * workPerThread < N && i < workPerThread; i++)
    {
        auto col = i + baseIdx * workPerThread;
        getSums<float>(input + col, sumCols[i], psumCols[i], M, N);
        const float mean = sumCols[i] / M;
        const float stdDevs = sqrtf(psumCols[i] / M - mean * mean);
        // Here 7 is a magic number(a.k.a. hyper-parameter)
        const float min = mean - 7 * stdDevs, max = mean + 7 * stdDevs;
        const float range = max - min;

        scales[col] = range / 255;
        const auto invScale = 255.0f / (range + kEpsilon);
        zeroPoints[col] = int32_t(std::nearbyintf(127 - max * invScale));

        int32_t sum = 0;

#pragma unroll
        for (std::size_t row = 0; row < M; ++row)
        {
            baseOutput[row * N + i] = climp<float, int8_t>(std::nearbyintf(input[col + row * N] * invScale + zeroPoints[col]));
            sum += baseOutput[row * N + i];
        }
        sums[col] = sum;
    }
}

__global__ void dequantFloatMatrix(
    const int32_t *__restrict__ input,
    const int M,
    const int N,
    const int K,
    const int32_t *zeroPointsA, const int32_t *zeroPointsB,
    const float *scalesA, const float *scalesB,
    const int32_t *sumsA, const int32_t *sumsB,
    float *__restrict__ output)
{
    const int64_t baseIdx = (int)blockIdx.x * blockDim.x;
    const auto tid = threadIdx.x;
    const auto tx = tid / 16, ty = tid % 16;
    const size_t baseX = 16 * blockIdx.x, baseY = 16 * blockIdx.y;
    const auto warpId = tid / 32;
    __shared__ int32_t sumA[16], sumB[16], zeroPointB[16];
    __shared__ float scaleB[16];

    if (baseX < M && baseY < N)
    {
        const auto scaleA = *scalesA; // broadcast here
        const auto zeroPointA = *zeroPointsA;
        if (warpId == 0) // warp 0 to copy
        {
            if (tid < 16 && baseX + tid < M)
                sumA[tid] = sumsA[baseX + tid];
            else if (tid >= 16 && baseY + tid - 16 < N)
            {
                zeroPointB[tid - 16] = zeroPointsB[baseY + tid - 16];
                scaleB[tid - 16] = scalesB[baseY + tid - 16];
                sumB[tid - 16] = sumsB[baseY + tid - 16];
            }
        }
        __syncthreads();
        if (baseX + tx < M && baseY + ty < N)
        {
            auto baseA = input[(baseX + tx) * N + baseY + ty];

            output[(baseX + tx) * N + baseY + ty] = scaleA * scaleB[ty] * (baseA - zeroPointA * sumB[ty] - zeroPointB[ty] * sumA[tx] + K * zeroPointA * zeroPointB[ty]);
        }
    }
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, hipblasHandle_t handle, float alpha = 1, float beta = 0)
{
    constexpr int workPerThread = 2;
    constexpr int threadsPerBlockSize = 256;
    dim3 threadsPerBlock(threadsPerBlockSize);
    dim3 numInputBlocks((M + threadsPerBlock.x * workPerThread - 1) / threadsPerBlock.x * workPerThread);
    dim3 numWeightBlocks((N + threadsPerBlock.x * workPerThread - 1) / threadsPerBlock.x * workPerThread);
    dim3 numDequantBlocks((M + 15) / 16, (N + 15) / 16);
    static thread_local quantMatrixHolder quantA(M, K, MIN_MAX), quantB(K, N, PER_COL);

    static int32_t *quantC = nullptr;
    static size_t Csize = 0;

    quantA.resize(M, K);
    quantB.resize(K, N);

    if (Csize * 2 < M * N)
    {
        hipFree(quantC);
        hipMalloc(&quantC, sizeof(int32_t) * M * N * 2);
        Csize = M * N;
    }

    void *d_temp_storage = nullptr;
    size_t size, prev_size = 0;
    static float *min = nullptr, *max = nullptr;

    if (min == nullptr)
    {
        hipMalloc(&min, sizeof(float));
        hipMalloc(&max, sizeof(float));
    }

    hipcub::DeviceReduce::Max(nullptr, size, a, max, M * K);
    if (size > prev_size * 2)
    {
        hipFree(d_temp_storage);
        hipMalloc(&d_temp_storage, size * 2);
        prev_size = size;
    }

    hipcub::DeviceReduce::Max(d_temp_storage, size, a, max, M * K); // These two can be fused into one.
    hipcub::DeviceReduce::Min(d_temp_storage, size, a, min, M * K);

#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    quantInput<threadsPerBlockSize, workPerThread><<<numInputBlocks, threadsPerBlock>>>(a, M, K, max, min, quantA.dataPtr(), *quantA.scalesPtr(), *quantA.zeroPointsPtr(), quantA.sumsPtr());
    quantWeight<threadsPerBlockSize, workPerThread><<<numWeightBlocks, threadsPerBlock>>>(b, K, N, quantB.dataPtr(), quantB.scalesPtr(), quantB.zeroPointsPtr(), quantB.sumsPtr());
#endif
    int32_t i32alpha = alpha, i32beta = beta;
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 N, M, K,
                 &i32alpha,
                 quantB.dataPtr(), HIP_R_8I, N,
                 quantA.dataPtr(), HIP_R_8I, K,
                 &i32beta,
                 quantC, HIP_R_32I, N,
                 HIPBLAS_COMPUTE_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    dequantFloatMatrix<<<numDequantBlocks, threadsPerBlock>>>(quantC, M, N, K, quantA.zeroPointsPtr(), quantB.zeroPointsPtr(), quantA.scalesPtr(), quantB.scalesPtr(), quantA.sumsPtr(), quantB.sumsPtr(), c);
#endif
}
