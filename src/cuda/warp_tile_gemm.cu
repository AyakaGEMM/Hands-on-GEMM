#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#ifndef __CUDACC__
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
void __syncthreads(); // workaround __syncthreads warning
void __syncwarp();
#endif
#include <iostream>
constexpr size_t BLOCK_SIZE = 16; // we assume that every block has equal blockDim.x and blockDim.y
constexpr size_t BLOCK_M = 128;   // These const values decide how many thing a thread compute and the amount of shared memory to allocate.
constexpr size_t BLOCK_N = 128;
constexpr size_t BLOCK_K = 8; // don't set 64 here, it will cause bank conflict and lower occupancy.
constexpr size_t BLOCK_M_COMPUTE = BLOCK_M / BLOCK_SIZE;
constexpr size_t BLOCK_N_COMPUTE = BLOCK_N / BLOCK_SIZE;

constexpr int shared_memory_A = BLOCK_M * BLOCK_K;
constexpr int shared_memory_B = BLOCK_N * BLOCK_K;
constexpr int shared_memory_element = shared_memory_A + shared_memory_B;
constexpr int shared_memory_size = shared_memory_element * sizeof(float); // shared memory to use.
#define colM(a, i, j, lda) a[((j) * (lda)) + (i)]
#define rowM(a, i, j, lda) a[(j) + (i) * (lda)]

constexpr __forceinline__ __device__ auto convertColIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subM, n = idx % subM;
    return begin + m + n * N;
}

constexpr __forceinline__ __device__ auto convertRowIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subN, n = idx % subN;
    return begin + m * N + n;
}

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    const size_t baseX = blockIdx.x * blockDim.x * BLOCK_M_COMPUTE;
    const size_t baseY = blockIdx.y * blockDim.y * BLOCK_N_COMPUTE;

    const int moveNum = shared_memory_element / (BLOCK_SIZE * BLOCK_SIZE) / 2;
    const size_t baseIdx = threadIdx.y * blockDim.y + threadIdx.x;

    constexpr size_t threadsNum = BLOCK_SIZE * BLOCK_SIZE;

    float c[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};
    float resC[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};

    __shared__ float subA[BLOCK_M * BLOCK_K];
    __shared__ float subB[BLOCK_N * BLOCK_K];

    float4 regB[BLOCK_M_COMPUTE / 4]; // hopefully, these should reside in register.
    float4 regA[BLOCK_M_COMPUTE / 4];

    const float *baseA = A + baseX * K;
    const float *baseB = B + baseY;

    int colA = baseIdx / 2, colB = baseIdx / (BLOCK_N / 4), rowA = (baseIdx & 1) * 4, rowB = (baseIdx * 4) % BLOCK_N;
    int warpId = baseIdx / 32, warpBaseId = baseIdx % 32;
    int colC = (warpId / 2 * 4 + warpBaseId % 4) * BLOCK_M_COMPUTE, rowC = ((warpId % 2) * 8 + warpBaseId / 4) * BLOCK_N_COMPUTE;
    float *baseC = C + (baseX + colC) * N + baseY + rowC;

    for (int i = 0; i < K; i += BLOCK_K)
    {
        regB[0] = *reinterpret_cast<const float4 *>(baseB + i * N + colB * N + rowB);
        regA[0] = *reinterpret_cast<const float4 *>(baseA + i + colA * K + rowA);
        *reinterpret_cast<float4 *>(&subB[baseIdx * 4]) = regB[0];
        subA[colA + rowA * BLOCK_M] = regA[0].x;
        subA[colA + (rowA + 1) * BLOCK_M] = regA[0].y;
        subA[colA + (rowA + 2) * BLOCK_M] = regA[0].z;
        subA[colA + (rowA + 3) * BLOCK_M] = regA[0].w;

        __syncthreads();
#pragma unroll
        for (int ii = 0; ii < BLOCK_K; ii++)
        {
            regA[0] = *reinterpret_cast<float4 *>(&subA[colC + ii * BLOCK_M]);
            regA[1] = *reinterpret_cast<float4 *>(&subA[(colC + 4) + ii * BLOCK_M]);

            regB[0] = *reinterpret_cast<float4 *>(&subB[rowC + BLOCK_N * ii]);
            regB[1] = *reinterpret_cast<float4 *>(&subB[rowC + 4 + BLOCK_N * ii]);

#pragma unroll
            for (int cpi = 0; cpi < BLOCK_M_COMPUTE / 4; cpi++)
            {
#pragma unroll
                for (int cpj = 0; cpj < BLOCK_N_COMPUTE / 4; cpj++)
                {
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].x * regB[cpj].x;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].x * regB[cpj].y;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].x * regB[cpj].z;
                    c[cpi * 4 * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].x * regB[cpj].w;

                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].y * regB[cpj].x;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].y * regB[cpj].y;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].y * regB[cpj].z;
                    c[(cpi * 4 + 1) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].y * regB[cpj].w;

                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].z * regB[cpj].x;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].z * regB[cpj].y;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].z * regB[cpj].z;
                    c[(cpi * 4 + 2) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].z * regB[cpj].w;

                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4] += regA[cpi].w * regB[cpj].x;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 1] += regA[cpi].w * regB[cpj].y;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 2] += regA[cpi].w * regB[cpj].z;
                    c[(cpi * 4 + 3) * BLOCK_M_COMPUTE + cpj * 4 + 3] += regA[cpi].w * regB[cpj].w;
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j += 4)
            *reinterpret_cast<float4 *>(&resC[i * BLOCK_M_COMPUTE + j]) = *reinterpret_cast<float4 *>(&baseC[i * N + j]);

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j++)
            resC[i * BLOCK_M_COMPUTE + j] = resC[i * BLOCK_M_COMPUTE + j] * beta + alpha * c[i * BLOCK_M_COMPUTE + j];

#pragma unroll
    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
#pragma unroll
        for (int j = 0; j < BLOCK_N_COMPUTE; j += 4)
            *reinterpret_cast<float4 *>(&baseC[i * N + j]) = *reinterpret_cast<float4 *>(&resC[i * BLOCK_M_COMPUTE + j]);
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}
