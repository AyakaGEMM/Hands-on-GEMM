#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#ifndef __CUDACC__
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
void __syncthreads(); // workaround __syncthreads warning
#endif
#include <iostream>
#define BLOCK_SIZE 16 // we assume that every block has equal blockDim.x and blockDim.y

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    int baseX = blockIdx.x * blockDim.x;
    int baseY = blockIdx.y * blockDim.y;

    float c = 0;

    if (tx < M && ty < N)
    {
        for (int i = 0; i < K; i++)
        {
            c += A[tx * K + i] * B[i * N + ty];
        }
        C[tx * N + ty] = beta * C[tx * N + ty] + alpha * c; // we multiply alpha here to reduce the alpha cal num.
    }
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / (threadsPerBlock.x), (N + threadsPerBlock.y - 1) / (threadsPerBlock.y));
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}
