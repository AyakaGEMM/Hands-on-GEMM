#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <mma.h>
using namespace nvcuda;
constexpr int WMMA_M = 16;
constexpr int WMMA_N = 16;
constexpr int WMMA_K = 16;
constexpr int BLOCK_M = 256;
constexpr int BLOCK_N = 128;
constexpr int BLOCK_K = 64;
#include <iostream>

__global__ void i8gemm256x128x64(const int8_t *A, const int8_t *B, int32_t *C,
                                 int M, int N, int K, const int32_t alpha, const int32_t beta)
{
    const int lda = K;
    const int ldb = K;
    const int ldc = N;

    constexpr int sharedLda = 64;
    constexpr int sharedLdb = 64;

    const size_t baseIdx = threadIdx.x;

    const auto warpM = (baseIdx / 32) / 4;
    const auto warpN = (baseIdx / 32) % 4;
    const auto laneId = baseIdx % 32;
    const auto warpId = baseIdx / 32;

    const auto baseA = A + blockIdx.x * BLOCK_M * lda;
    const auto baseB = B + blockIdx.y * BLOCK_N * ldb;
    const auto baseC = C + blockIdx.x * BLOCK_M * ldc + blockIdx.y * BLOCK_N + (warpId / 2) * 64 * ldc + (warpId & 1) * 64;

    __shared__ int8_t shared_mem[BLOCK_M * sharedLda + BLOCK_N * sharedLdb];
    auto sharedA = shared_mem;
    auto sharedB = shared_mem + BLOCK_M * sharedLda;

    int32_t frag_c[64][2] = {}, frag_d[64][2] = {}; // Initialize to 0.
    int32_t frag_a[8], frag_b[8];                   // Use streaming read to release the reg pressure.

#pragma unroll
    for (int k = 0; k < K; k += BLOCK_K)
    {
// Do 64x64x64 (mnk) mma at a time according to cutlass.
#pragma unroll
        for (int i = 0; i < 16; i++)
        {
            *reinterpret_cast<int32_t *>(&sharedA[(baseIdx / 16 + i * 16) * sharedLda + (baseIdx % 16) * 4]) = *reinterpret_cast<const int32_t *>(&baseA[(baseIdx / 16 + i * 16) * lda + (baseIdx % 16) * 4 + k]);
        }

// Need transpose here, I leave it here for now.
#pragma unroll
        for (int i = 0; i < 8; i++)
        {
            *reinterpret_cast<int32_t *>(&sharedB[(baseIdx / 16 + i * 16) * sharedLdb + (baseIdx % 16) * 4]) = *reinterpret_cast<const int32_t *>(&baseB[(baseIdx / 16 + i * 16) * ldb + (baseIdx % 16) * 4 + k]);
        }

        __syncthreads();
        // Load matrix in 4 stages, could try warp shuff and overlap in the future.

#pragma unroll
        for (int ik = 0; ik < 4; ik++)
        {
#pragma unroll
            for (int i = 0; i < 8; i++)
            {
                frag_b[i] = *reinterpret_cast<int32_t *>(&sharedB[((warpId & 1) * 64 + i * 8 + laneId / 4) * sharedLdb + (laneId % 4) * 4 + ik * 16]);
                frag_a[i] = *reinterpret_cast<int32_t *>(&sharedA[((warpId / 2) * 64 + i * 8 + laneId / 4) * sharedLda + (laneId % 4) * 4 + ik * 16]);
            }

#pragma unroll
            for (int im = 0; im < 8; im++)
            {
#pragma unroll
                for (int in = 0; in < 8; in++)
                {
                    asm(
                        "mma.sync.aligned.m8n8k16.row.col.s32.s8.s8.s32 \
                        {%0, %1}, \
                        {%2}, {%3}, \
                        {%0, %1};"
                        : "+r"(frag_c[im * 8 + in][0]), "+r"(frag_c[im * 8 + in][1])
                        : "r"(frag_a[im]), "r"(frag_b[in])); // With an implicit __syncwarp() here.
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (int im = 0; im < 8; im++)
    {
#pragma unroll
        for (int in = 0; in < 8; in++)
        {
            auto idx = im * 8 + in;
            *reinterpret_cast<int64_t *>(frag_d[idx]) = *reinterpret_cast<int64_t *>(&baseC[(im * 8 + laneId / 4) * ldc + in * 8 + (laneId & 3) * 2]); // I'm the reinterpret_cast master!
            frag_d[idx][0] = frag_d[idx][0] * beta + frag_c[idx][0] * alpha;
            frag_d[idx][1] = frag_d[idx][1] * beta + frag_c[idx][1] * alpha;
            *reinterpret_cast<int64_t *>(&baseC[(im * 8 + laneId / 4) * ldc + in * 8 + (laneId & 3) * 2]) = *reinterpret_cast<int64_t *>(frag_d[idx]);
        }
    }
}

void i8gemm(int M, int N, int K, int8_t *a, int8_t *b, int32_t *c, int32_t alpha, int32_t beta)
{
    dim3 threadsPerBlock(256);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
    i8gemm256x128x64<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
}