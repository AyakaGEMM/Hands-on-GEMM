#include <cstdlib>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#ifndef __CUDACC__
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
void __syncthreads(); // workaround __syncthreads warning
#endif
#include <iostream>
#define BLOCK_COMPUTE 4
const size_t BLOCK_SIZE = 16; // we assume that every block has equal blockDim.x and blockDim.y
const size_t BLOCK_M = 64;    // These const values decide how many thing a thread compute and the amount of shared memory to allocate.
const size_t BLOCK_N = 64;
const size_t BLOCK_K = 16; // don't set 64 here, it will cause bank conflict and lower occupancy.
const size_t BLOCK_M_COMPUTE = BLOCK_M / BLOCK_SIZE;
const size_t BLOCK_N_COMPUTE = BLOCK_N / BLOCK_SIZE;
const size_t BLOCK_K_COMPUTE = BLOCK_K / BLOCK_SIZE;

const int shared_memory_A = BLOCK_M * BLOCK_K;
const int shared_memory_B = BLOCK_N * BLOCK_K;
const int shared_memory_element = shared_memory_A + shared_memory_B;
const int shared_memory_size = shared_memory_element * sizeof(float); // shared memory to use.
#define colM(a, i, j, lda) a[((j) * (lda)) + (i)]
#define rowM(a, i, j, lda) a[(j) + (i) * (lda)]

__forceinline__ __device__ float convertColIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subM, n = idx % subM;
    return begin[m + n * N];
}

__forceinline__ __device__ float convertRowIdx(int idx, const float *begin, int subM, int subN, int N)
{
    int m = idx / subN, n = idx % subN;
    return begin[m * N + n];
}

__global__ void matrixMul(const float *A, const float *B, float *C,
                          int M, int N, int K, float alpha, float beta)
{
    int tx = blockIdx.x * blockDim.x * BLOCK_COMPUTE + threadIdx.x;
    int ty = blockIdx.y * blockDim.y * BLOCK_COMPUTE + threadIdx.y;

    int baseX = blockIdx.x * blockDim.x * BLOCK_M_COMPUTE;
    int baseY = blockIdx.y * blockDim.y * BLOCK_N_COMPUTE;

    const int moveNum = shared_memory_element / (BLOCK_SIZE * BLOCK_SIZE);
    const float *baseA = A + baseX * K + baseY;
    // const float *baseB = B;

    float c[BLOCK_M_COMPUTE * BLOCK_N_COMPUTE] = {};

    __shared__ float subA[BLOCK_SIZE * BLOCK_SIZE * BLOCK_M_COMPUTE * BLOCK_K_COMPUTE];
    __shared__ float subB[BLOCK_SIZE * BLOCK_SIZE * BLOCK_N_COMPUTE * BLOCK_K_COMPUTE];
    for (int i = 0; i < K; i += BLOCK_K)
    {
        for (int idx = (threadIdx.x * blockDim.x + threadIdx.y) * moveNum; idx < (threadIdx.x * blockDim.x + threadIdx.y) * moveNum + moveNum; idx++)
        {
            if (idx < shared_memory_A)
                subA[idx] = convertColIdx(idx, A + baseX * K + i, BLOCK_M, BLOCK_K, K);
            else
                subB[idx - shared_memory_A] = convertRowIdx(idx - shared_memory_A, B + baseY + i * N, BLOCK_K, BLOCK_N, N);
        }
        // for (int cpi = 0; cpi < BLOCK_COMPUTE; cpi++)
        //     for (int cpj = 0; cpj < BLOCK_COMPUTE; cpj++)
        //     {
        //         colM(subA, threadIdx.x + cpi * BLOCK_SIZE, threadIdx.y + cpj * BLOCK_SIZE, blockDim.x * BLOCK_COMPUTE) = A[K * (tx + cpi * BLOCK_SIZE) + i + threadIdx.y + cpj * BLOCK_SIZE]; // Here we use col major to store smemA to avoid cache miss.
        //         subB[blockDim.y * BLOCK_COMPUTE * (threadIdx.x + cpi * BLOCK_SIZE) + (threadIdx.y + cpj * BLOCK_SIZE)] = B[N * (i + threadIdx.x + cpi * BLOCK_SIZE) + ty + cpj * BLOCK_SIZE];
        //     }
        __syncthreads();
#pragma unroll(4)
        for (int ii = 0; ii < BLOCK_K; ii++)
        {
            float regB[BLOCK_M_COMPUTE]; // hopefully, these should reside in register.
#pragma unroll
            for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
            {
                regB[cpj] = subB[threadIdx.y * BLOCK_N_COMPUTE + cpj + BLOCK_N * ii];
            }
#pragma unroll
            for (int cpi = 0; cpi < BLOCK_M_COMPUTE; cpi++)
            {
                float regA = subA[(threadIdx.x * BLOCK_M_COMPUTE + cpi) + ii * BLOCK_M];
#pragma unroll
                for (int cpj = 0; cpj < BLOCK_N_COMPUTE; cpj++)
                {
                    c[cpi * BLOCK_M_COMPUTE + cpj] += regA * regB[cpj];
                }
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < BLOCK_M_COMPUTE; i++)
        for (int j = 0; j < BLOCK_N_COMPUTE; j++)
            C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] = beta * C[(baseX + threadIdx.x * BLOCK_M_COMPUTE + i) * N + baseY + threadIdx.y * BLOCK_N_COMPUTE + j] + alpha * c[i * BLOCK_M_COMPUTE + j];
}

void sgemm(int M, int N, int K, float *a, float *b, float *c, float alpha = 1, float beta = 0)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((M + BLOCK_M - 1) / BLOCK_M, (N + BLOCK_N - 1) / BLOCK_N);
#ifdef __HIPCC__ // workaround for stupid vscode intellisense
    matrixMul<<<numBlocks, threadsPerBlock>>>(a, b, c, M, N, K, alpha, beta);
#endif
}
