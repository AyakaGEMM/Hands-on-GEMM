#include "hip/hip_runtime.h"
// Copied from https://github.com/Cjkkkk/CUDA_gemm/blob/14b517370609d322647c55fe9136b6d81c2ba9a7/benchmark/benchmark_dense.cu

#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuda_help_func.hpp"
#include "utils.hpp"

#define ASIZE(type) (sizeof(type) * M * K)
#define BSIZE(type) (sizeof(type) * K * N)
#define CSIZE(type) (sizeof(type) * M * N)

extern void sgemm(int, int, int, float *, float *, float *, bool beta = false);

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    float *h_A = (float *)malloc(ASIZE(float));
    float *h_B = (float *)malloc(BSIZE(float));
    float *h_C = (float *)malloc(CSIZE(float));
    float *h_C1 = (float *)malloc(CSIZE(float));

    float *d_A;
    float *d_B;
    float *d_C;

    checkCudaErrors(hipMalloc(&d_A, ASIZE(float)));
    checkCudaErrors(hipMalloc(&d_B, BSIZE(float)));
    checkCudaErrors(hipMalloc(&d_C, CSIZE(float)));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 96;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 64;
    const int THREAD_SIZE_Y = 6;
    const int THREAD_SIZE_X = 4;
    const bool ENABLE_DOUBLE_BUFFER = false;

    float alpha = 2.0;
    float beta = 2.0;

    // 生成A的数据
    genRandomMatrix(h_A, M, K);
    genRandomMatrix(h_B, K, N);
    genRandomMatrix(h_C, M, N);
    copyMatrix(h_C1, h_C, M, N);

    checkCudaErrors(hipMemcpy(d_A, h_A, ASIZE(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, BSIZE(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    checkCudaErrors(hipMemcpy(d_C, h_C, CSIZE(float), hipMemcpyHostToDevice));

    dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    if (N % BLOCK_SIZE_N != 0)
        dimGrid.x++;
    if (M % BLOCK_SIZE_M != 0)
        dimGrid.y++;

    // warm up here
    sgemm(M, N, K, d_A, d_B, d_C);
    checkCudaErrors(hipEventRecord(start));

    printf("Grid Dim: (%d %d) Block Dim: (%d %d)\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
    for (int run = 0; run < nIter; run++)
    {
        sgemm(M, N, K, d_A, d_B, d_C);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy(h_C, d_C, CSIZE(float), hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
           gigaFlops[0],
           msecPerMatrixMul[0],
           flopsPerMatrixMul);

    // cublas
    hipblasHandle_t blas_handle;
    checkCuBlasErrors(hipblasCreate(&blas_handle));
    checkCudaErrors(hipMemcpy(d_C, h_C1, CSIZE(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++)
    {
        checkCuBlasErrors(
            hipblasSgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        M, N, K, &alpha,
                        d_A, K, d_B, N, &beta, d_C, N));
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy(h_C1, d_C, CSIZE(float), hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
           gigaFlops[1],
           msecPerMatrixMul[1],
           flopsPerMatrixMul);

    hipblasDestroy(blas_handle);

    double eps = 1.e-6; // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++)
    {
        double abs_err = fabs(h_C[i] - h_C1[i]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], h_C1[i], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}