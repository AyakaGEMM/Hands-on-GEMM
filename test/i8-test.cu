#include <stdio.h>
#include <stdlib.h>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuda_help_func.hpp"
#include "utils.hpp"

#define ASIZE(type) (sizeof(type) * M * K)
#define BSIZE(type) (sizeof(type) * K * N)
#define CSIZE(type) (sizeof(type) * M * N)

int MAX;

extern void i8gemm(int, int, int, int8_t *, int8_t *, int32_t *, int32_t, int32_t);

void refgemm(int M, int N, int K, int8_t *a, int8_t *b, int32_t *c, int32_t alpha, int32_t beta)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            c[i * N + j] *= beta;
            for (int k = 0; k < K; k++)
            {
                c[i * N + j] += alpha * a[i * K + k] * b[k * N + j];
            }
        }
    }
}

int main(int argc, char **argv)
{
    // if (argc != 4)
    //{
    //     printf("usage: ./main [M] [K] [N]\n");
    //     exit(0);
    // }
    size_t M = 256;
    size_t N = 256;
    size_t K = 256;

    std::cout << M << " " << N << " " << K << std::endl;

    int8_t *h_A = new int8_t[M * M];
    int8_t *h_B = new int8_t[M * M];
    int32_t *h_C = new int32_t[M * M];
    int32_t *h_C1 = new int32_t[M * M];

    int8_t *d_A;
    int8_t *d_B;
    int32_t *d_C;

    checkCudaErrors(hipMalloc(&d_A, ASIZE(int8_t)));
    checkCudaErrors(hipMalloc(&d_B, BSIZE(int8_t)));
    checkCudaErrors(hipMalloc(&d_C, CSIZE(int32_t)));

    std::cout << d_C << std::endl;

    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    constexpr int32_t alpha = 1;
    constexpr int32_t beta = 0;

    // 生成A的数据
    genRandomMatrix(h_A, M, K);
    genRandomMatrix(h_B, K, N);
    genRandomMatrix(h_C, M, N);
    copyMatrix(h_C1, h_C, M, N);

    checkCudaErrors(hipMemcpy(d_A, h_A, ASIZE(int8_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, BSIZE(int8_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, h_C, CSIZE(int32_t), hipMemcpyHostToDevice)); // Free Memory

    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++)
        {
            h_C[i * N + j] = beta * h_C[i * N + j];
            for (int k = 0; k < K; k++)
                h_C[i * N + j] += alpha * h_A[i * K + k] * h_B[k * N + j];
        }
    // showMatrix(h_C, M, N, "Matrix Ref");
    copyMatrix(h_C, h_C1, M, N);

    hipEvent_t start, stop;
    hipblasHandle_t blas_handle;
    checkCuBlasErrors(hipblasCreate(&blas_handle));
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    i8gemm(M, N, K, d_A, d_B, d_C, alpha, beta);
    checkCudaErrors(hipMemcpy(h_C, d_C, CSIZE(int32_t), hipMemcpyDeviceToHost));
    checkCuBlasErrors(
        hipblasGemmEx(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     N, M, K,
                     &alpha,
                     d_B, HIP_R_8I, N,
                     d_A, HIP_R_8I, K,
                     &beta,
                     d_C, HIP_R_32I, N,
                     HIPBLAS_COMPUTE_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    checkCudaErrors(hipMemcpy(h_C1, d_C, CSIZE(int32_t), hipMemcpyDeviceToHost));

    showMatrix(h_A, M, K, "Matrix A");
    showMatrix(h_B, K, N, "Matrix B");
    showMatrix(h_C, M, N, "Matrix C1");
    showMatrix(h_C1, M, N, "Matrix C2");

    i8gemm(M, N, K, d_A, d_B, d_C, alpha, beta);
    checkCudaErrors(hipMemcpy(h_C, d_C, CSIZE(int32_t), hipMemcpyDeviceToHost));
    showMatrix(h_C, M, N, "Matrix C1");

    refgemm(M, N, K, h_A, h_B, h_C, 1, -1);
    showMatrix(h_C, M, N, "Matrix C3");

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    delete[] h_A;
    printf("Ok A\n");
    delete[] h_B;
    printf("Ok B\n");
    delete[] h_C;
    printf("Ok C\n");
    delete[] h_C1;
    printf("Ok C1\n");
}